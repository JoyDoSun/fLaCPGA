#include "hip/hip_runtime.h"
/* flac in cuda */
#include <getopt.h>
#include<ctype.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <vector>

#include "wavereader.hpp"
#include "bitwriter.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Should probably window data also...
__global__ void calculate_lags(
    const float *const dpcm_buf, float * dlags, 
    const int spb, const int64_t samples){
/* Calculates lags and leaves them in memory like so:
 * |b0i0l0,b0i1l0...b0i4095l0|b0i0l1,b0i1l1...b0i4095l1|
 * ...
 * |b1i0l0,b1i1l0...
 *
 * Launch this kernel like so:
 * <<<(1 << 12, 4, 12),1024>>>
 * <<<dim3(blocks, breakup of block within grid, lags), 
 * maximum threads/block>>> */    

    int t = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int j = blockIdx.z;
    int N = blockDim.x; 
    int n = blockDim.y;
    int l = blockDim.z;
    int i = t + by*n + bx*N;
    
    float a = dpcm_buf[i];
    float b = dpcm_buf[i + j];
    
    dlags[i + j*4096] = a*b;
}

// Reduce the autocorrelation lags...
__global__ void sum_lags(const float * const dlags, float * dslags, const int spb, const int64_t samples){
    
    
}

__global__ void levinson_durbinson(const float * const dautoc, float * dmodelc, const int spb, const int64_t samples){
    
    
}

/*
 * Stage 1:
 * data[i]*modelc[i] 
 */
__global__ void fir_stage1(const float * const dpcm_buf, const float * const dmodelc, 
                           float *dfir_s1, const int spb, const int64_t samples){
    int j; // Which model coefficient
    int i; // Which data part
    dfir_s1[i+j] = dpcm_buf[i+j]*modelc[i+j];
    
}

/* Stage 2:
 * Sum the data from stage 1 to get result
 */
__global__ void fir_stage2(const float * const dfir_s1, float *dfir_s2, const int spb,
                           const int64_t samples){
}

/* Find best model
 * Check error in each model, select best one
 */
__global__ void fbm_stage1(const float * const dpcm_buf, const float * const dfir_s2, 
                           float *error, const int spb, const int64_t samples){
    
    /* Calculate error */
    int i;
    error[i] = dpcm_buf[i] - dfir_s2[i];
    
    /* Sum error in each block and model */
    
    /* Choose lowest error in each model */
    
    
    
}

/* 
 * Do rice encoding of error
 */

__global__ void fbr_stage1(const float * const error, float * tbpb, const int spb, const int64_t samples){
    /* encode each error using param */
    int param;
    int lmask;
    int i;
    
    tbpb[i] = error[i] >> param + (error[i] & lmask);
    
    /* do prefix sums of the errors for each param */
    
    /* Pick the lowest summed error */
}

/* 
 * Using the prefix sum errors, each thread can place
 * its rice encoded error in the correct place in the final output
 * Need to also be doing crc checks and then storing framing data........ :(
 */


void exit_with_help(char *argv[]){
    fprintf(stderr, "usage: %s [ OPTIONS ] infile.wav [outfile.flac]\n", argv[0]);
    exit(1);
}


int main(int argc, char *argv[]){
    int opt = 0, fixed = 0, verbatim = 0, single = 0, encode = 0;
    int order = 0;
    while ((opt = getopt(argc,argv,"fvs:")) != EOF)
        switch(opt)
        {
            case 'f': fixed = 1; encode = 1; break;
            case 'v': verbatim = 1; encode = 1; break;
            case 's': single = 1; 
                      encode = 1; 
                      std::cerr << "OPTARG::: " << optarg << "\n";
                      if (strcmp(optarg, "v") == 0)
                          verbatim = 1;
                      else
                        order = atoi(optarg);
                      break;
            case 'h':
            case '?': 
            default:
                exit_with_help(argv);
        }
        
    std::shared_ptr<std::fstream> fin;
    std::shared_ptr<std::fstream> fout;
    
    
    if (optind < argc){
        fin = std::make_shared<std::fstream>(argv[optind], std::ios::in | std::ios::binary);
        if(fin->fail()) {
            fprintf(stderr, "ERROR: opening %s for input\n", argv[optind]);
            return 1;
        }
    }
    
    auto fr = std::make_shared<BitReader>(fin);
    auto wr = std::make_shared<WaveReader>();
    wr->read_metadata(fr);
    auto meta = wr->getMetaData();
    meta->print(stdout);
    
    const int samples = 1 << 22; // ~1MB of samples
    
    int16_t * hpcm_buf = (int16_t *) malloc(sizeof(int16_t) * samples);
    
    float * hpcmf_buf = (float *) malloc(sizeof(float) * samples);
    
    float * dpcm_buf;
    hipMalloc(&dpcm_buf, sizeof(float) * samples);
    
    gpuErrchk(hipPeekAtLastError());
    auto tsamples = meta->getNumSamples();
    
    /*if (tsamples < samples){
        wr->read_data(fr, hpcm_buf, tsamples);
    } else {
        */
    tsamples -= wr->read_data(fr, hpcm_buf, samples);
    for (int i = 0; i < samples; i++) {
        hpcmf_buf[i] = (float) hpcm_buf[i];
    }
    

    
    float *dlags;
    hipMalloc(&dlags, sizeof(float)*samples*12);
    gpuErrchk(hipPeekAtLastError());
    
    float *hlags = (float *) malloc(sizeof(float)*4096);
    
    
    hipMemcpyAsync(dpcm_buf, hpcmf_buf, samples, hipMemcpyHostToDevice, 0);
    gpuErrchk(hipPeekAtLastError());
    
    calculate_lags<<<dim3(1 << 12, 4, 12),1024>>>(dpcm_buf, dlags, 4096, samples);
    gpuErrchk(hipPeekAtLastError());
    
    hipDeviceSynchronize();
    hipMemcpy(hlags, dlags, 4096, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 60; i++){
        printf("i: %d :: %d %f %f\n", i, hpcm_buf[i], hpcmf_buf[i], hlags[i]);
    }
    
    hipDeviceSynchronize();
    
    return 0;
}
